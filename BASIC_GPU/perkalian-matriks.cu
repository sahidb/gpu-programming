/*
 * Tutorial CUDA - Perkalian Matriks
 * ================================================================
 * Dibuat oleh : I Wayan Aditya Swardiana
 * Email       : iway020@brin.go.id
 * ================================================================
 * Perkalian 2 matriks A (m x n) * B (n x o) = C (m x o)
 * Matriks direpresentasikan dalam row-major format
 * Dikompilasi dan dites pada CUDA 10.2 dan gcc 7.3.0 pada HPC BRIN
 * Kompilasi dengan perintah:
        $ module load cuda/10
        $ module load gcc/7
 *      $ nvcc perkalian-matriks.cu -o mat.o
 * Jalankan dengan perintah:
 *      $ module load cuda/10
 *      $ ./mat.o nilai_m nilai_n nilai_o
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// kernel CUDA untuk perkalian matriks secara simpel
// 1 block mengerjakan 1 baris (m)
// tiap block memiliki thread sejumlah kolom (o)
// tiap thread menghitung perkalian matriks untuk baris m & kolom o
__global__ void gpu_perkalian_matriks_simpel(int *matriks_a, int *matriks_b, int *matriks_c, int m, int n, int o)
{
    int indeks_baris = blockIdx.x;
    int indeks_kolom = threadIdx.x;
    int jumlah = 0;

    if (indeks_baris < m && indeks_kolom < o)
    {
        for (int i = 0; i < n; i++)
        {
            jumlah += matriks_a[indeks_baris * n + i] * matriks_b[i * o + indeks_kolom];
        }
        matriks_c[indeks_baris * o + indeks_kolom] = jumlah;
    }
}

__global__ void gpu_perkalian_matriks_threadblock_2d(int *matriks_a, int *matriks_b, int *matriks_c, int m, int n, int o)
{

}

__global__ void gpu_perkalian_matriks_shared_memory(int *matriks_a, int *matriks_b, int *matriks_c, int m, int n, int o)
{

}

// fungsi untuk perkalian matriks secara simpel di CPU
void cpu_perkalian_matriks_simpel(int *matriks_a, int *matriks_b, int *matriks_c, int m, int n, int o)
{
    for (int indeks_baris = 0; indeks_baris < m; indeks_baris++)
    {
        for (int indeks_kolom = 0; indeks_kolom < o; indeks_kolom++)
        {
            int jumlah = 0;
            for (int i = 0; i < n; i++)
            {
                jumlah += matriks_a[indeks_baris * n + i] * matriks_b[i * o + indeks_kolom];
            }
            matriks_c[indeks_baris * o + indeks_kolom] = jumlah;
        }
    }
}

// fungsi untuk inisialisasi matriks secara random
void cpu_inisialisasi_matriks(int *matriks, int jumlah_baris, int jumlah_kolom)
{
    for (int indeks_baris = 0; indeks_baris < jumlah_baris; indeks_baris++)
    {
        for (int indeks_kolom = 0; indeks_kolom < jumlah_kolom; indeks_kolom++)
        {
            matriks[indeks_baris * jumlah_kolom + indeks_kolom] = rand() % 10;
        }
    }
}

// fungsi untuk mencetak matriks
void cpu_print_matriks(int *matriks, int jumlah_baris, int jumlah_kolom)
{
    for (int indeks_baris = 0; indeks_baris < jumlah_baris; indeks_baris++)
    {
        printf("[ ");
        for (int indeks_kolom = 0; indeks_kolom < jumlah_kolom; indeks_kolom++)
        {
            printf("%d ", matriks[indeks_baris * jumlah_kolom + indeks_kolom]);
        }
        printf("]\n");
    }
}

// fungsi untuk membandingkan hasil perkalian matriks di CPU & GPU
void cpu_validasi_hasil(int *matriks_cpu, int *matriks_gpu, int jumlah_baris, int jumlah_kolom)
{
    bool cek_hasil = true;
    for (int indeks_baris = 0; indeks_baris < jumlah_baris; indeks_baris++)
    {
        for (int indeks_kolom = 0; indeks_kolom < jumlah_kolom; indeks_kolom++)
        {
            if (matriks_cpu[indeks_baris * jumlah_kolom + indeks_kolom] != matriks_gpu[indeks_baris * jumlah_kolom + indeks_kolom])
            {
                cek_hasil = false;
            }
        }
    }

    if(cek_hasil)
    {
        printf("Hasil perkalian matriks di CPU dan GPU sama.\n");
    }
    else
    {
        printf("Hasil perkalian matriks di CPU dan GPU tidak sama.\n");
    }
}

int main(int argc, char const *argv[])
{
    // inisialisasi nilai m, n, dan o
    int m = atoi(argv[1]); 
    int n = atoi(argv[2]);
    int o = atoi(argv[3]);
   
    printf("PERKALIAN MATRIKS - MATRIKS A (%d x %d) * MATRIKS B (%d x %d)\n", m, n, n, o);
    printf("===============================\n");

    // inisialisasi matriks di host (CPU)
    int *host_matriks_a, *host_matriks_b, *host_matriks_c_cpu, *host_matriks_c_gpu;

    host_matriks_a = (int *) malloc (sizeof(int) * m * n);
    host_matriks_b = (int *) malloc (sizeof(int) * n * n);
    host_matriks_c_cpu = (int *) malloc (sizeof(int) * m * o);
    host_matriks_c_gpu = (int *) malloc (sizeof(int) * m * o);

    cpu_inisialisasi_matriks(host_matriks_a, m, n);
    cpu_inisialisasi_matriks(host_matriks_b, n, o);

    printf("Matriks A (%d x %d)\n", m, n);
    cpu_print_matriks(host_matriks_a, m, n);
    printf("===============================\n");
    printf("Matriks B (%d x %d)\n", n, o);
    cpu_print_matriks(host_matriks_b, n, o);
    printf("===============================\n");

    // eksekusi fungsi perkalian matriks di CPU
    cpu_perkalian_matriks_simpel(host_matriks_a, host_matriks_b, host_matriks_c_cpu, m, n, o);

    printf("Matriks C CPU (%d x %d)\n", m, o);
    cpu_print_matriks(host_matriks_c_cpu, m, o);
    printf("===============================\n");

    // inisialisasi matriks di device (GPU)
    int *device_matriks_a, *device_matriks_b, *device_matriks_c;

    hipMalloc((void **) &device_matriks_a, sizeof(int) * m * n);
    hipMalloc((void **) &device_matriks_b, sizeof(int) * n * o);
    hipMalloc((void **) &device_matriks_c, sizeof(int) * m * o);

    // salin matriks input dari host ke device
    hipMemcpy(device_matriks_a, host_matriks_a, sizeof(int) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(device_matriks_b, host_matriks_b, sizeof(int) * n * o, hipMemcpyHostToDevice);

    // inisialisasi thread block
    dim3 jumlah_block(m, 1, 1);
    dim3 jumlah_thread_per_block(o, 1, 1);

    // eksekusi kernel CUDA perkalian matriks di GPU
    gpu_perkalian_matriks_simpel<<<jumlah_block, jumlah_thread_per_block>>>(device_matriks_a, device_matriks_b, device_matriks_c, m, n, o);

    // salin matriks hasil dari device ke host
    hipMemcpy(host_matriks_c_gpu, device_matriks_c, sizeof(int) * m * o, hipMemcpyDeviceToHost);

    printf("Matriks C GPU (%d x %d)\n", m, o);
    cpu_print_matriks(host_matriks_c_gpu, m, o);
    printf("===============================\n");

    // cek hasil perkalian matriks
    cpu_validasi_hasil(host_matriks_c_cpu, host_matriks_c_gpu, m, o);
    
    // bersihkan memory device
    hipFree(device_matriks_a);
    hipFree(device_matriks_b);
    hipFree(device_matriks_c);
}